
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void my_kernel() {
        printf("Hello from block %i of %i and thread %i \n ", blockIdx.x, blockDim.x, threadIdx.x);

}

int main() {
    my_kernel <<<16, 16 >>> ();
    hipError_t cuda_err = hipDeviceSynchronize();
    if (cuda_err != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cuda_err));
    return 0;
}

