#include "hip/hip_runtime.h"
#include "ZWEngine.h"
#include <iostream>

// settings
const unsigned int SCR_WIDTH = 800;
const unsigned int SCR_HEIGHT = 600;

const GLchar *vs_shader_path = "../glsl/vertex_shader.glsl";
const GLchar *fs_shader_path = "../glsl/fragment_shader.glsl";


__global__ void add(int n, float *x, float *y) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}
int main() {
    auto *tmp_app = new ZWEngine();

    if (!tmp_app->init_engine(SCR_WIDTH, SCR_HEIGHT)) {
        std::cout << "engine failed to initialize" << std::endl;
    } else {
        std::cout << "engine initialized successfully" << std::endl;
    }
    tmp_app->init_shader_program(vs_shader_path, fs_shader_path);
    std::cout << "engine start running" << std::endl;
    tmp_app->run();
}